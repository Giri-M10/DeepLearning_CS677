#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
//#include <hip/hip_runtime_api.h>
#include "dot.h"

// Main function
int main(int argc ,char* argv[]) {


FILE *data;
FILE *weights;
size_t size;
	

// Declaring the rows and columns and CUDA device and number of threads by taking input from the user 
unsigned int rows=atoi(argv[3]);
unsigned int cols=atoi(argv[4]);
int CUDA_DEVICE = atoi(argv[5]);
int THREADS = atoi(argv[6]);
printf("Put in values\n");
printf("Rows= %d\n,Cols = %d\n,CUDA_DEVICE= %d\n, THREADS =%d \n",rows,cols,CUDA_DEVICE,THREADS);
hipError_t err = hipSetDevice(CUDA_DEVICE);
if(err != hipSuccess) { printf("Error in setting the CUDA device\n"); exit(EXIT_FAILURE); }



// Declaring the variable for the host
int BLOCKS;
float* host_results = (float*) malloc(rows * sizeof(float)); 
struct timeval starttime, endtime;
clock_t start, end;
float seconds = 0;
unsigned int jobs; 
unsigned long i;



// Declaring the variable for the devices
float  *dev_dataT;
float *dev_dataV;
float *results;
//size_t len = 0;
float arr[rows][cols];
float var ;
int vrow =1;
start = clock();



// Validate if the file is readable
data = fopen(argv[1], "r");
weights = fopen(argv[2],"r");
if (data == NULL) {
  printf("Error in reading in the data");
	return 0;
}
if (weights == NULL){
	printf("Error in reading in the weights");
}
size = (size_t)((size_t)rows*(size_t)cols);
size_t sizeV = 0;
sizeV = (size_t)((size_t)vrow*(size_t)cols);
//printf("The size of the data is %lu\n",size);
fflush(stdout);


// Memory allocation for the dat files
float *dataT = (float*)malloc((size)*sizeof(float));
float *dataV = (float*)malloc((sizeV)*sizeof(float));
if(dataT == NULL) {
	printf("Error in allocating memory for the data file.\n");
}
if(dataV == NULL){
	printf("Error in allocating memory for the weights file. \n");
}  
gettimeofday(&starttime, NULL);

int j = 0;
// Moving the data from the file to the allocated memory
for (i =0; i< rows;i++){
	for(j=0; j<cols ; j++){
		fscanf(data,"%f",&var);
      arr[i][j]=var;
}
}
for (i =0;i<cols;i++){
  for(j= 0; j<rows; j++){
		dataT[rows*i+j]= arr[j][i];
}
}		
for (j=0;j<cols;j++){
	fscanf(weights,"%f",&dataV[j]);
}   
fclose(data);
fclose(weights);
printf("Reading data has completed........\n");
fflush(stdout);
gettimeofday(&endtime, NULL);
seconds+=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
printf("Time for reading the data %f\n", seconds);




// Memory allocation on the GPU for the data
printf("Calculating time for memory allocation on GPU for the data");
gettimeofday(&starttime, NULL);
err = hipMalloc((float**) &dev_dataT, (size_t) size * (size_t) sizeof(float));
if(err != hipSuccess) { printf("Error in allocating memory on the GPU\n"); }
gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
printf("Time for memory allocation of the data %f\n", seconds);
gettimeofday(&starttime, NULL);


// test vector
err = hipMalloc((float**) &dev_dataV, sizeV * sizeof(float));
if(err != hipSuccess) { printf("Error in allocating memory on GPU\n"); }
gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
printf("The time it took to allocate memory for the weights is %f\n", seconds);
gettimeofday(&starttime, NULL);
	
// Memory allocation on GPU
printf("Calculating time for memory allocation on GPU for testvector");
err = hipMalloc((float**) &results, rows * sizeof(float) );
if(err != hipSuccess) { printf("Error in allocating memory on the GPU for the results\n"); }
gettimeofday(&endtime, NULL); 
seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
printf("time for cudamalloc for result =%f\n", seconds);

// Copying the data to the GPU
printf("Currently copying the data to the allocated memory in GPU\n");
gettimeofday(&starttime, NULL);
err = hipMemcpy(dev_dataT, dataT, (size_t)size *sizeof(float), hipMemcpyHostToDevice);
if(err != hipSuccess) { printf("Error in copying data to the GPU\n"); }
gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
printf("Time for copying the data to the GPU %f\n", seconds);

// Copying weights to the allocated memory on the GPU
gettimeofday(&starttime, NULL);
err = hipMemcpy(dev_dataV, dataV, sizeV*sizeof(float), hipMemcpyHostToDevice);
if(err != hipSuccess) { printf("Error in copying the weights to the GPU\n"); }
gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
printf("The time it took to copy the weights to the GPU is %f\n", seconds);
jobs = rows;
BLOCKS = (jobs + THREADS - 1)/THREADS;
gettimeofday(&starttime, NULL);

// calling  the kernel function
kernel<<<BLOCKS,THREADS>>>(rows,cols,dev_dataT,	dev_dataV, results);
        gettimeofday(&endtime, NULL); seconds=((double)endtime.tv_sec+(double)endtime.tv_usec/1000000)-((double)starttime.tv_sec+(double)starttime.tv_usec/1000000);
printf("The time it took for the kernel is %f\n", seconds);
		
// copy the results back to the CPU
hipMemcpy(host_results,results,rows * sizeof(float),hipMemcpyDeviceToHost);
printf("The dot product of the data and weights is \n");
printf("\n");

for(int k = 0; k < jobs; k++) {
	printf("%f ", host_results[k]);
	printf("\n");
}
printf("\n");
hipFree( dev_dataT );
hipFree( results );
end = clock();
seconds = (float)(end - start) / CLOCKS_PER_SEC;
printf("Time it took to run the whole program is %f\n", seconds);
return 0;
}
